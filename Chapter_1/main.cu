
#include <hip/hip_runtime.h>
#include <iostream>


using namespace std;

__global__ void add(int a, int b, int *c) {
    *c = a + b;

}

int main() {
    int c(32);
    int *dev_c;
    hipError_t alloc = hipMalloc((void**)&dev_c, sizeof(int));
    if (alloc != hipSuccess) {
        cerr << "Impossible to allocate memory: " << hipGetErrorString(alloc) << endl;
        exit(EXIT_FAILURE);
    }
    add<<<1, 1>>>(2, 7, dev_c);

    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    cout << "2 + 7 = " << c << endl;

    hipFree(dev_c);

    cout << "CUDA" << endl;

    int count;
    hipGetDeviceCount(&count);
    cout << "nb CUDA compatible devices: " << count << endl;

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    cout << "Properties" << endl;
    cout << "major: " << prop.major << endl;
    cout << "minor: " << prop.minor << endl;
    cout << "texture max dim: " << prop.maxTexture2D[0] << " " << prop.maxTexture2D[1] << endl;
    cout << "nb multiprocessors: " << prop.multiProcessorCount << endl;
    cout << "max threads per block " << prop.maxThreadsPerBlock << endl;
    cout << "max thread dim: " << prop.maxThreadsDim[0] << " " << prop.maxThreadsDim[1] << "  " << prop.maxThreadsDim[2] << endl;
    cout << "max grid dim: " << prop.maxGridSize[0] << " " << prop.maxGridSize[1] << "  " << prop.maxGridSize[2] << endl;

    cout << "device name: " << prop.name << endl;
    cout << "total global mem " << prop.totalGlobalMem << endl;
    cout << "max shared mem per block: " << prop.sharedMemPerBlock << endl;
    cout << "nb registers per block: " << prop.regsPerBlock << endl;
    cout << "nb thread in a warp: " << prop.warpSize << endl;



    return 0;
}
