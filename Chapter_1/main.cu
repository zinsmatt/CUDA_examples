
#include <hip/hip_runtime.h>
#include <iostream>


using namespace std;

__global__ void add(int a, int b, int *c) {
    *c = a + b;

}

int main() {
    int c(32);
    int *dev_c;
    hipError_t alloc = hipMalloc((void**)&dev_c, sizeof(int));
    if (alloc != hipSuccess) {
        cerr << "Impossible to allocate memory: " << hipGetErrorString(alloc) << endl;
        exit(EXIT_FAILURE);
    }
    add<<<1, 1>>>(2, 7, dev_c);

    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    cout << "2 + 7 = " << c << endl;

    hipFree(dev_c);

    cout << "CUDA" << endl;

    return 0;
}
