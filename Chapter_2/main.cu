
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

using namespace std;

__global__ void add_vector(int* a, int* b, int* c, int size)
{
    int id = blockIdx.x * 96 + threadIdx.x;
    if (id<size)
    {
        c[id] = a[id] + b[id];
    }
}


int main()
{
    int v1, v2;
    cout << "First value ?  " << endl;
    cin >> v1;
    cout << "Second value ? " << endl;
    cin >> v2;
    cout << "v1 = " << v1 << endl;
    cout << "v2 = " << v2 << endl;
    const int size = 20000;


    int *a = new int[size];
    int *b = new int[size];
    int *c = new int[size];

    for (int i=0; i<size; ++i)
    {
        a[i] = v1;
        b[i] = v2;
    }

    int *a_dev, *b_dev, *c_dev;

    if (hipSuccess != hipMalloc((void**)&a_dev, sizeof(int) * size))
    {
        cerr << "error allocation a_dev" << endl;
    }
    if (hipSuccess != hipMalloc((void**)&b_dev, sizeof(int) * size))
    {
        cerr << "error allocation b_dev" << endl;
    }
    if (hipSuccess != hipMalloc((void**)&c_dev, sizeof(int) * size))
    {
        cerr << "error allocation c_dev" << endl;
    }

    if (hipSuccess != hipMemcpy(a_dev, a, sizeof(int) * size, hipMemcpyHostToDevice))
    {
        cerr << "error cuda mem copy" << endl;
    }

    if (hipSuccess != hipMemcpy(b_dev, b, sizeof(int) * size, hipMemcpyHostToDevice))
    {
        cerr << "error cuda mem copy" << endl;
    }


    // dim3 block_dim(1, 1, 1);
    // dim3 threads_per_block(size, 1, 1);
    int nb_blocks = size / 96;

    add_vector<<<nb_blocks+1, 96>>>(a_dev, b_dev, c_dev, size);

    if (hipSuccess != hipMemcpy(c, c_dev, sizeof(int) * size, hipMemcpyDeviceToHost))
    {
        cerr << "error cuda mem copy back" << endl;
    }

    for (int i=0; i<size; ++i)
    {
        cout << c[i] << " ";
    }
    cout << endl;

    delete[] a;
    delete[] b;
    delete[] c;
    hipFree(a_dev);
    hipFree(b_dev);
    hipFree(c_dev);
    return 0;
}
